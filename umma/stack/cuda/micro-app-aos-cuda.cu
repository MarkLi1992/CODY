#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <getopt.h>

#include <hip/hip_runtime.h>
#include "micro-app-cuda.h"

struct edge edges[NEDGES];
float pt_data[NPOINTS * 3];
float edge_data[NEDGES];

void print_help() {
    printf("Usage: \n");
    printf("\t --help print this message and exit \n");
    printf("\t --type Type of graph, must be one of:\n");
    printf("\t\t\t pure_random \n");
    printf("\t\t\t regular_random \n");
    printf("\t\t\t contiguous \n");
    printf("\t\t\t file \n");
    printf("\t --nloops Number of repetitions, must be \n");
    printf("\t          at least one. \n");
    printf("\t --file File from which to read graph \n");
}

double timer() {
    struct timeval tp;
    struct timezone tzp;

    gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec) + ((double) tp.tv_usec) * 1e-6;
}

int data_init() {
    int i;

    for (i = 0; i < NPOINTS; i++) {
        pt_data[3*i+0] = 1;
        pt_data[3*i+1] = 1;
        pt_data[3*i+2] = 1;
    }

    return 0;
}

int edge_data_init() {
    int i;

    for (i = 0; i < NEDGES; i++) {
        edge_data[i] = 1;
    }

    return 0;
}

__global__ void edge_gather(float* pt_data, float* edge_data,
        struct edge* edges, int nedges) {

    int i;
    int v0;
    int v1;

    i = blockIdx.x * NTHREADS + threadIdx.x;
    if (i < nedges) {
        v0 = edges[i].v0;
        v1 = edges[i].v1;

        edges[i].v0_pt_data[0] = pt_data[3*v0+0];
        edges[i].v0_pt_data[1] = pt_data[3*v0+1];
        edges[i].v0_pt_data[2] = pt_data[3*v0+2];

        edges[i].v1_pt_data[0] = pt_data[3*v1+0];
        edges[i].v1_pt_data[1] = pt_data[3*v1+1];
        edges[i].v1_pt_data[2] = pt_data[3*v1+2];
        

        edges[i].data = edge_data[i];
    }
}

__global__ void edge_compute(struct edge* edges, int nedges) {
    int i;
    float v0_p0, v0_p1, v0_p2;
    float v1_p0, v1_p1, v1_p2;
    float x0, x1, x2;
    float e_data;

    i = blockIdx.x * NTHREADS + threadIdx.x;

    if (i < nedges) {
        v0_p0 = edges[i].v0_pt_data[0];
        v0_p1 = edges[i].v0_pt_data[1];
        v0_p2 = edges[i].v0_pt_data[2];

        v1_p0 = edges[i].v1_pt_data[0];
        v1_p1 = edges[i].v1_pt_data[1];
        v1_p2 = edges[i].v1_pt_data[2];

        e_data = edges[i].data;

        x0 = (v0_p0 + v1_p0) * e_data;
        x1 = (v0_p1 + v1_p1) * e_data;
        x2 = (v0_p2 + v1_p2) * e_data;

        edges[i].v0_pt_data[0] = x0;
        edges[i].v0_pt_data[1] = x1;
        edges[i].v0_pt_data[2] = x2;

        edges[i].v1_pt_data[0] = x0;
        edges[i].v1_pt_data[1] = x1;
        edges[i].v1_pt_data[2] = x2;
    }
}

__global__ void edge_scatter(float* pt_data, struct edge* edges, 
        int nedges) {
    int i;
    int v0;
    int v1;

    i = blockIdx.x * NTHREADS + threadIdx.x;
       
    if (i < nedges) {
        v0 = edges[i].v0;
        v1 = edges[i].v1;

        atomicAdd(&pt_data[3*v0+0], edges[i].v0_pt_data[0]);
        atomicAdd(&pt_data[3*v0+1], edges[i].v0_pt_data[1]);
        atomicAdd(&pt_data[3*v0+2], edges[i].v0_pt_data[2]);

        atomicAdd(&pt_data[3*v1+0], edges[i].v1_pt_data[0]);
        atomicAdd(&pt_data[3*v1+1], edges[i].v1_pt_data[1]);
        atomicAdd(&pt_data[3*v1+2], edges[i].v1_pt_data[2]);
    }
}

int main(int argc, char** argv) {
    int i;
    int rv;
    double time0, time1;
    int c, opt_i;
    int nloops = 0;
    char* gt = "";
    char* fname = "";

    float* d_pt_data;
    struct edge* d_edges;
    float* d_edge_data;

    int nBlocks = (NEDGES / NTHREADS) + 1;

    static struct option long_opts[] = {
        {"help",   no_argument,       0, 0},
        {"type",   required_argument, 0, 0},
        {"nloops", required_argument, 0, 0},
        {"file",   required_argument, 0, 0}
    };

    /* Parse command-line arguments */
    while (1) {
        c = getopt_long(argc, argv, "", 
                long_opts, &opt_i);

        if (c == -1) {
            break;
        }

        if (c == 0) {
            switch (opt_i) {
                case 0:
                    print_help();
                    exit(0);
                case 1:
                    gt = optarg;
                    break;
                case 2:
                    nloops = atoi(optarg);
                    break;
                case 3:
                    fname = optarg;
                    break;
            }
        } else {
            print_help();
            exit(0);
        }
    }

    /* check for errors */
    if (gt == NULL || nloops < 1) {
        print_help();
        exit(0);
    }

    // initialize data structures
    rv = graph_init_aos(gt, NPOINTS, NEDGES, edges, fname);
    if (rv < 0) {
        printf("Error creating graph. \n");
        exit(0);
    }

    data_init();
    edge_data_init();

    // allocate memory on the GPU
    hipMalloc((void**) &d_pt_data, NPOINTS * 3 * sizeof(float));
    hipMalloc((void**) &d_edges, NEDGES * sizeof(struct edge));
    hipMalloc((void**) &d_edge_data, NEDGES * sizeof(float));

    // loop
    time0 = timer();
    for (i = 0; i < nloops; i++) {

        /* 
         * Edge Gather
         */
        // copy over 
        hipMemcpy(d_pt_data, pt_data, NPOINTS * 3 * sizeof(float),
                hipMemcpyHostToDevice);
        hipMemcpy(d_edges, edges, NEDGES * sizeof(struct edge),
                hipMemcpyHostToDevice);
        hipMemcpy(d_edge_data, edge_data, NEDGES * sizeof(float),
                hipMemcpyHostToDevice);

        // invoke kernel
        edge_gather<<<nBlocks,NTHREADS>>>(d_pt_data, d_edge_data, d_edges, NEDGES);

        // copy back
        hipMemcpy(edges, d_edges, NEDGES * sizeof(struct edge),
                hipMemcpyDeviceToHost);

        /*
         * Edge Compute
         */
        // copy over
        hipMemcpy(d_edges, edges, NEDGES * sizeof(struct edge),
                hipMemcpyHostToDevice);

        // call kernel
        edge_compute<<<nBlocks,NTHREADS>>>(d_edges, NEDGES);
        
        // copy back
        hipMemcpy(edges, d_edges, NEDGES * sizeof(struct edge),
                hipMemcpyDeviceToHost);

        /* 
         * Edge Scatter
         */
        // copy over 
        hipMemcpy(d_pt_data, pt_data, NPOINTS * 3 * sizeof(float),
                hipMemcpyHostToDevice);
        hipMemcpy(d_edges, edges, NEDGES * sizeof(struct edge),
                hipMemcpyHostToDevice);

        // call kernel
        edge_scatter<<<nBlocks,NTHREADS>>>(d_pt_data, d_edges, NEDGES);
        
        // copy back
        hipMemcpy(pt_data, d_pt_data, NPOINTS * 3 * sizeof(float),
                hipMemcpyDeviceToHost);

    }
    time1 = timer();

    // free memory
    hipFree(d_pt_data);
    hipFree(d_edges);
    hipFree(d_edge_data);

    // print results
    for (i = 0; i < 10; i++) {
        printf("%i : %f %f %f \n", i, pt_data[3*i+0], 
                pt_data[3*i+1], pt_data[3*i+2]);
    }

    printf("Time: %f s \n", (time1 - time0) / ((float) nloops));

    return 0;
}
